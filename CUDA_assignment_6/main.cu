#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <time.h>

extern "C"
{
#include "libs/bitmap.h"
}

#define cudaErrorCheck(ans)               \
  {                                       \
    gpuAssert((ans), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5

int sobelYFilter[] = {-1, -2, -1,
                      0, 0, 0,
                      1, 2, 1};

int sobelXFilter[] = {-1, -0, 1,
                      -2, 0, 2,
                      -1, 0, 1};

int laplacian1Filter[] = {-1, -4, -1,
                          -4, 20, -4,
                          -1, -4, -1};

int laplacian2Filter[] = {0, 1, 0,
                          1, -4, 1,
                          0, 1, 0};

int laplacian3Filter[] = {-1, -1, -1,
                          -1, 8, -1,
                          -1, -1, -1};

int gaussianFilter[] = {1, 4, 6, 4, 1,
                        4, 16, 24, 16, 4,
                        6, 24, 36, 24, 6,
                        4, 16, 24, 16, 4,
                        1, 4, 6, 4, 1};

const char *filterNames[] = {"SobelY", "SobelX", "Laplacian 1", "Laplacian 2", "Laplacian 3", "Gaussian"};
int *const filters[] = {sobelYFilter, sobelXFilter, laplacian1Filter, laplacian2Filter, laplacian3Filter, gaussianFilter};
unsigned int const filterDims[] = {3, 3, 3, 3, 3, 5};
float const filterFactors[] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0 / 256.0};

int const maxFilterIndex = sizeof(filterDims) / sizeof(unsigned int);

void cleanup(char **input, char **output)
{
  if (*input)
    free(*input);
  if (*output)
    free(*output);
}

void graceful_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(0);
}

void error_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(1);
}

// Helper function to swap bmpImageChannel pointers

void swapImageRawdata(pixel **one, pixel **two)
{
  pixel *helper = *two;
  *two = *one;
  *one = helper;
}

void swapImage(bmpImage **one, bmpImage **two)
{
  bmpImage *helper = *two;
  *two = *one;
  *one = helper;
}

// Apply convolutional filter on image data
void applyFilter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++)
  {
    for (unsigned int x = 0; x < width; x++)
    {
      int ar = 0, ag = 0, ab = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++)
      {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++)
        {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
          {
            ar += in[yy * width + xx].r * filter[nky * filterDim + nkx];
            ag += in[yy * width + xx].g * filter[nky * filterDim + nkx];
            ab += in[yy * width + xx].b * filter[nky * filterDim + nkx];
          }
        }
      }

      ar *= filterFactor;
      ag *= filterFactor;
      ab *= filterFactor;

      ar = (ar < 0) ? 0 : ar;
      ag = (ag < 0) ? 0 : ag;
      ab = (ab < 0) ? 0 : ab;

      out[y * width + x].r = (ar > 255) ? 255 : ar;
      out[y * width + x].g = (ag > 255) ? 255 : ag;
      out[y * width + x].b = (ab > 255) ? 255 : ab;
    }
  }
}

// Apply convolutional filter on image data
__global__ void applyFilter_CUDA_Kernel(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{
  unsigned int const filterCenter = (filterDim / 2);
  for (unsigned int y = 0; y < height; y++)
  {
    for (unsigned int x = 0; x < width; x++)
    {
      int ar = 0, ag = 0, ab = 0;
      for (unsigned int ky = 0; ky < filterDim; ky++)
      {
        int nky = filterDim - 1 - ky;
        for (unsigned int kx = 0; kx < filterDim; kx++)
        {
          int nkx = filterDim - 1 - kx;

          int yy = y + (ky - filterCenter);
          int xx = x + (kx - filterCenter);
          if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
          {
            ar += in[yy * width + xx].r * filter[nky * filterDim + nkx];
            ag += in[yy * width + xx].g * filter[nky * filterDim + nkx];
            ab += in[yy * width + xx].b * filter[nky * filterDim + nkx];
          }
        }
      }

      ar *= filterFactor;
      ag *= filterFactor;
      ab *= filterFactor;

      ar = (ar < 0) ? 0 : ar;
      ag = (ag < 0) ? 0 : ag;
      ab = (ab < 0) ? 0 : ab;

      out[y * width + x].r = (ar > 255) ? 255 : ar;
      out[y * width + x].g = (ag > 255) ? 255 : ag;
      out[y * width + x].b = (ab > 255) ? 255 : ab;
    }
  }
}

void help(char const *exec, char const opt, char const *optarg)
{
  FILE *out = stdout;
  if (opt != 0)
  {
    out = stderr;
    if (optarg)
    {
      fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
    }
    else
    {
      fprintf(out, "Invalid parameter - %c\n", opt);
    }
  }
  fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
  fprintf(out, "\n");
  fprintf(out, "Options:\n");
  fprintf(out, "  -k, --filter     <filter>        filter index (0<=x<=%u) (2)\n", maxFilterIndex - 1);
  fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

  fprintf(out, "\n");
  fprintf(out, "Example: %s before.bmp after.bmp -i 10000\n", exec);
}

int main(int argc, char **argv)
{
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;
  unsigned int filterIndex = 2;

  static struct option const long_options[] = {
      {"help", no_argument, 0, 'h'},
      {"filter", required_argument, 0, 'k'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}};

  static char const *short_options = "hk:i:";
  {
    char *endptr;
    int c;
    int parse;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1)
    {
      switch (c)
      {
      case 'h':
        help(argv[0], 0, NULL);
        graceful_exit(&input, &output);
      case 'k':
        parse = strtol(optarg, &endptr, 10);
        if (endptr == optarg || parse < 0 || parse >= maxFilterIndex)
        {
          help(argv[0], c, optarg);
          error_exit(&input, &output);
        }
        filterIndex = (unsigned int)parse;
        break;
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg)
        {
          help(argv[0], c, optarg);
          error_exit(&input, &output);
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind + 1))
  {
    help(argv[0], ' ', "Not enough arugments");
    error_exit(&input, &output);
  }

  unsigned int arglen = strlen(argv[optind]);
  input = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(input, argv[optind], arglen);
  optind++;

  arglen = strlen(argv[optind]);
  output = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(output, argv[optind], arglen);
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0, 0);
  if (image == NULL)
  {
    fprintf(stderr, "Could not allocate new image!\n");
    error_exit(&input, &output);
  }

  if (loadBmpImage(image, input) != 0)
  {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    error_exit(&input, &output);
  }

  printf("Apply filter '%s' on image with %u x %u pixels for %u iterations\n", filterNames[filterIndex], image->width, image->height, iterations);

  // Time measurement init
  hipEvent_t start_time, end_time;
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);

  // Here we do the actual computation!
  // image->data is a 2-dimensional array of pixel which is accessed row first ([y][x])
  // image->rawdata is a 1-dimensional array of pixel containing the same data as image->data
  // each pixel is a struct of 3 unsigned char for the red, blue and green colour channel
  // bmpImage *processImage = newBmpImage(image->width, image->height);

  // TODO: Cuda malloc and memcpy the rawdata from the images, from host side to device side
  int image_size = image->width * image->height * sizeof(pixel);
  int filter_size = filterDims[filterIndex] * filterDims[filterIndex] * sizeof(int);

  pixel *d_image_rawdata, *d_process_image_rawdata;
  int *d_filter;

  hipMalloc((void **)&d_image_rawdata, image_size);
  hipMalloc((void **)&d_process_image_rawdata, image_size);
  hipMalloc((void **)&d_filter, filter_size);

  hipMemcpy(d_image_rawdata, image->rawdata, image_size, hipMemcpyHostToDevice);
  hipMemcpy(d_filter, filters[filterIndex], filter_size, hipMemcpyHostToDevice);

  // ? Do we also need to copy the filters?
  // ? __device__ maybe

  // TODO: Define the gridSize and blockSize, e.g. using dim3 (see Section 2.2. in CUDA Programming Guide)

  // Start time measurement
  hipEventRecord(start_time);

  for (unsigned int i = 0; i < iterations; i++)
  {
    // TODO: Implement kernel call instead of serial implementation
    applyFilter_CUDA_Kernel<<<1, 1>>>(d_process_image_rawdata, // Out
                                      d_image_rawdata,         // In
                                      image->width,
                                      image->height,
                                      // filters[filterIndex],
                                      d_filter,
                                      filterDims[filterIndex],
                                      filterFactors[filterIndex]);
    // swapImage(&processImage, &image);
    swapImageRawdata(&d_process_image_rawdata, &d_image_rawdata);
  }

  // End time measurement
  hipEventRecord(end_time);

  hipMemcpy(image->rawdata, d_image_rawdata, image_size, hipMemcpyDeviceToHost);

  hipFree(d_image_rawdata);
  hipFree(d_process_image_rawdata);
  hipFree(d_filter);

  // Blocks CPU execution until end_time is recorded
  hipEventSynchronize(end_time);

  float spentTime = 0.0;
  hipEventElapsedTime(&spentTime, start_time, end_time);
  printf("Time spent: %.3f seconds\n", spentTime / 1000);

  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  // Check for error
  hipError_t error = hipPeekAtLastError();
  if (error)
  {
    fprintf(stderr, "A CUDA error has occurred while cracking: %s\n", hipGetErrorString(error));
  }

  //Write the image back to disk
  if (saveBmpImage(image, output) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", output);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  graceful_exit(&input, &output);
};
