#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_fp16.h>
#include <mma.h> // CUDA WMMA API

using namespace nvcuda;

extern "C"
{
#include "libs/bitmap.h"
}

#define cudaErrorCheck(ans)               \
  {                                       \
    gpuAssert((ans), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

// Define some error checking macros.
#define cudaErrCheck(stat)                     \
  {                                            \
    cudaErrCheck_((stat), __FILE__, __LINE__); \
  }
void cudaErrCheck_(hipError_t stat, const char *file, int line)
{
  if (stat != hipSuccess)
  {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}

#define WARP_SIZE 32
#define BLOCK_DIMENSION 16 // A thread block size of 16x16 (256 threads) is a common choice (from https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy)

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5

int sobelYFilter[] = {-1, -2, -1,
                      0, 0, 0,
                      1, 2, 1};

int sobelXFilter[] = {-1, -0, 1,
                      -2, 0, 2,
                      -1, 0, 1};

int laplacian1Filter[] = {-1, -4, -1,
                          -4, 20, -4,
                          -1, -4, -1};

int laplacian2Filter[] = {0, 1, 0,
                          1, -4, 1,
                          0, 1, 0};

int laplacian3Filter[] = {-1, -1, -1,
                          -1, 8, -1,
                          -1, -1, -1};

int gaussianFilter[] = {1, 4, 6, 4, 1,
                        4, 16, 24, 16, 4,
                        6, 24, 36, 24, 6,
                        4, 16, 24, 16, 4,
                        1, 4, 6, 4, 1};

const char *filterNames[] = {"SobelY", "SobelX", "Laplacian 1", "Laplacian 2", "Laplacian 3", "Gaussian"};
int *const filters[] = {sobelYFilter, sobelXFilter, laplacian1Filter, laplacian2Filter, laplacian3Filter, gaussianFilter};
unsigned int const filterDims[] = {3, 3, 3, 3, 3, 5};
// float const filterFactors[] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0 / 256.0}; // Not used in this case

const unsigned int MAX_GRID_DIMENSION = 65535;

// const unsigned int numberOfChannels = 3;

// Hardcoded selected filters for now
const unsigned int numberOfFiltersUsed = 5;
const unsigned int filterIndexes[] = {0, 1, 2, 3, 4};
const unsigned int filterDim = 3; // Only one filter dimension supported for now

// WMMA stuff from https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/
// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// mxk * kxn = mxn
const unsigned int DESIRED_M = numberOfFiltersUsed; // 5 filters
const unsigned int DESIRED_K = 9;                   // 3x3 filters
const unsigned int DESIRED_N = 4000 * 2334;         // 4000x2334 image

// Must be multiplies of 16
const int MATRIX_M = 16;        // I want 5 here (5 filters)
const int MATRIX_K = 16;        // I want 9 here (3x3 filter values)
const int MATRIX_N = DESIRED_N; // It is evenly divisible by 16

// im2col and col2im taken from https://github.com/pluskid/Mocha.jl/blob/master/deps/im2col.cpp#L7
// The function works on one channel at the time because the output image should not be only one channel.
// Furthemore, we assume that we use a valid padding (input dim = output dim) and 1 stride.
void im2col(pixel *img, unsigned char *col, int width, int height, int filterDim, int color)
{
  // The dimension of the col array is im_width*im_height*filter_h*filter_w
  int kernel_h = filterDim,  // Assuming square kernel
      kernel_w = filterDim,  // Assuming square kernel
      pad_h = filterDim / 2, // Same padding
      pad_w = filterDim / 2; // Same padding
  int height_col = (height + 2 * pad_h - kernel_h) + 1;
  int width_col = (width + 2 * pad_w - kernel_w) + 1;
  int channels_col = kernel_h * kernel_w;

  for (int c = 0; c < channels_col; ++c)
  {
    int w_offset = c % kernel_w;
    int h_offset = (c / kernel_w) % kernel_h;
    int c_im = c / (kernel_h * kernel_w);

    for (int h = 0; h < height_col; ++h)
    {
      for (int w = 0; w < width_col; ++w)
      {
        int h_pad = h - pad_h + h_offset;
        int w_pad = w - pad_w + w_offset;
        int index_col = (c * height_col + h) * width_col + w;
        int index_im = (c_im * height + h_pad) * width + w_pad;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
        {
          if (color == 0)
          {
            col[index_col] = img[index_im].r;
          }
          else if (color == 1)
          {
            col[index_col] = img[index_im].g;
          }
          else
          {
            col[index_col] = img[index_im].b;
          }
        }
        else
        {
          col[index_col] = 0;
        }
      }
    }
  }
}

void buildFilterArray(float *array)
{
  // The filter is of dimension mxk
  // The 0s padding is included in this function
  for (int i = 0; i < MATRIX_M; i++)
  {
    for (int j = 0; j < MATRIX_K; j++)
    {
      int arrayIndex = i * MATRIX_K + j;
      if (i < DESIRED_M && j < DESIRED_K)
      {
        unsigned int filterIndex = filterIndexes[i];
        array[arrayIndex] = (float)filters[filterIndex][j];
      }
      else
      {
        array[arrayIndex] = 0.0f;
      }
    }
  }
}

void buildImageArray(unsigned char *outputCol, unsigned char *inputCol, unsigned int intputColLength)
{
  // The image is of dimension kxn
  // This function adds the 5 0s to the colums
  memcpy(outputCol, inputCol, intputColLength * sizeof(unsigned char));
  for (int k = 0; k < MATRIX_K - DESIRED_K; k++)
  {
    for (int n = 0; n < MATRIX_N; n++)
    {
      outputCol[intputColLength + k * MATRIX_N + n] = 0;
    }
  }
}

void cleanup(char **input, char **output)
{
  if (*input)
    free(*input);
  if (*output)
    free(*output);
}

void graceful_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(0);
}

void error_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(1);
}

// Helper function to swap bmpImageChannel pointers

void swapImageRawdata(pixel **one, pixel **two)
{
  pixel *helper = *two;
  *two = *one;
  *one = helper;
}

void swapImage(bmpImage **one, bmpImage **two)
{
  bmpImage *helper = *two;
  *two = *one;
  *one = helper;
}

// This function is only used for creating the baseline images for filter 0-4
// Apply convolutional filter on image data with the use of shared memory
__global__ void apply_filter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{

  // Now instead of using the filter directly from global memory, we want to copy the filter to shared memory.
  // Dynamic shared memory because the filterDim is not known at compile time.

  // This one holds all of the data
  extern __shared__ int s[];

  int *shared_filter = s;                                                // Length of filterDim * filterDim
  pixel *shared_pixels = (pixel *)&shared_filter[filterDim * filterDim]; // Length of BLOCK_DIMENSION * BLOCK_DIMENSION

  for (int i = 0; i < filterDim * filterDim; i++)
  {
    shared_filter[i] = filter[i];
  }

  // Sync to make sure that all threads have completed the loads to shared memory
  __syncthreads();
  // Now we can use shared_filter!

  // Because shared memory is only shared between blocks, it makes sense to make the shared memory array for
  // the image as big as the block, since each thread in the block changes one pixel.

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Handle out of bounds
  if (x >= width || y >= height)
  {
    // __syncthreads(); // ? Needed? Think so, to avoid deadlock
    return;
  }

  // Set the position in the block to the correct value
  shared_pixels[threadIdx.y * BLOCK_DIMENSION + threadIdx.x] = in[y * width + x];

  // Sync to make sure that all threads have completed the loads to shared memory
  __syncthreads();
  // Now we can use shared_pixels!

  unsigned int const filterCenter = (filterDim / 2);
  int ar = 0, ag = 0, ab = 0;
  for (unsigned int ky = 0; ky < filterDim; ky++)
  {
    int nky = filterDim - 1 - ky;
    for (unsigned int kx = 0; kx < filterDim; kx++)
    {
      int nkx = filterDim - 1 - kx;

      int yy = y + (ky - filterCenter);
      int xx = x + (kx - filterCenter);

      // Now, since the edge threads needs pixels outside the block's shared memory,
      // we need to check its position.

      if (xx >= 0 && xx < BLOCK_DIMENSION && yy >= 0 && yy < BLOCK_DIMENSION)
      {
        ar += shared_pixels[yy * BLOCK_DIMENSION + xx].r * shared_filter[nky * filterDim + nkx];
        ag += shared_pixels[yy * BLOCK_DIMENSION + xx].g * shared_filter[nky * filterDim + nkx];
        ab += shared_pixels[yy * BLOCK_DIMENSION + xx].b * shared_filter[nky * filterDim + nkx];
      }
      // Else if the normal code from task 1-4
      else if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
      {
        ar += in[yy * width + xx].r * shared_filter[nky * filterDim + nkx];
        ag += in[yy * width + xx].g * shared_filter[nky * filterDim + nkx];
        ab += in[yy * width + xx].b * shared_filter[nky * filterDim + nkx];
      }
    }
  }

  ar *= filterFactor;
  ag *= filterFactor;
  ab *= filterFactor;

  ar = (ar < 0) ? 0 : ar;
  ag = (ag < 0) ? 0 : ag;
  ab = (ab < 0) ? 0 : ab;

  out[y * width + x].r = (ar > 255) ? 255 : ar;
  out[y * width + x].g = (ag > 255) ? 255 : ag;
  out[y * width + x].b = (ab > 255) ? 255 : ab;
}

// __global__ void apply_filter_GEMM(bmpImage *out, bmpImage *in, int *filters, int numberOfFilters, unsigned int filterDim, unsigned int filterSize, float filterFactor)
__global__ void apply_filter_GEMM(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta)
{
  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // From https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma-example
  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag; // This holds A*B
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  // Initialize the output to zero
  wmma::fill_fragment(acc_frag, 0.0f);

  // A*B
  // Loop over the K-dimension
  for (int i = 0; i < K; i += WMMA_K)
  {
    int aRow = warpM * WMMA_M;
    int aCol = i;
    int bRow = i;
    int bCol = warpN * WMMA_N;

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N)
    {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // AB + C
  // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;

  if (cRow < M && cCol < N)
  {
    wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

    for (int i = 0; i < c_frag.num_elements; i++)
    {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
  }
}

__host__ void floatToHalf(half *out, float *in, int n)
{
  for (int i = 0; i < n; i++)
  {
    out[i] = __float2half(in[i]);
  }
}

// __global__ void convertFp32ToFp16(half *out, float *in, int n)
// {
//   int idx = blockDim.x * blockIdx.x + threadIdx.x;
//   if (idx < n)
//   {
//     out[idx] = in[idx];
//   }
// }

void help(char const *exec, char const opt, char const *optarg)
{
  FILE *out = stdout;
  if (opt != 0)
  {
    out = stderr;
    if (optarg)
    {
      fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
    }
    else
    {
      fprintf(out, "Invalid parameter - %c\n", opt);
    }
  }
  fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
  fprintf(out, "\n");
  fprintf(out, "Options:\n");
  fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

  fprintf(out, "\n");
  fprintf(out, "Example: %s before.bmp after.bmp -i 10000\n", exec);
}

int main(int argc, char **argv)
{
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;

  static struct option const long_options[] = {
      {"help", no_argument, 0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}};

  static char const *short_options = "h:i:";
  {
    char *endptr;
    int c;
    // int parse;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1)
    {
      switch (c)
      {
      case 'h':
        help(argv[0], 0, NULL);
        graceful_exit(&input, &output);
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg)
        {
          help(argv[0], c, optarg);
          error_exit(&input, &output);
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind + 1))
  {
    help(argv[0], ' ', "Not enough arugments");
    error_exit(&input, &output);
  }

  unsigned int arglen = strlen(argv[optind]);
  input = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(input, argv[optind], arglen);
  optind++;

  arglen = strlen(argv[optind]);
  output = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(output, argv[optind], arglen);
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0, 0);
  if (image == NULL)
  {
    fprintf(stderr, "Could not allocate new image!\n");
    error_exit(&input, &output);
  }

  if (loadBmpImage(image, input) != 0)
  {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    error_exit(&input, &output);
  }

  // Const initialization
  for (unsigned int i = 1; i < numberOfFiltersUsed; i++)
  {
    if (filterDims[filterIndexes[i]] != filterDim)
    {
      printf("Unequal filter dimension used");
      exit(1);
    }
  }

  float *filterCol = (float *)malloc(MATRIX_M * MATRIX_K * sizeof(float));
  buildFilterArray(filterCol);

  unsigned int tempImageColLength = image->width * image->height * filterDim * filterDim;
  unsigned char *tempImageCol_r = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  unsigned char *tempImageCol_g = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  unsigned char *tempImageCol_b = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  im2col(image->rawdata, tempImageCol_r, image->width, image->height, filterDim, 0);
  im2col(image->rawdata, tempImageCol_g, image->width, image->height, filterDim, 1);
  im2col(image->rawdata, tempImageCol_b, image->width, image->height, filterDim, 2);

  unsigned char *imageColChar_r = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  unsigned char *imageColChar_g = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  unsigned char *imageColChar_b = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  buildImageArray(imageColChar_r, tempImageCol_r, tempImageColLength);
  buildImageArray(imageColChar_g, tempImageCol_g, tempImageColLength);
  buildImageArray(imageColChar_b, tempImageCol_b, tempImageColLength);

  // TOOD try to fix these, it crashes on free _b
  tempImageCol_r = NULL;
  tempImageCol_g = NULL;
  tempImageCol_b = NULL;
  free(tempImageCol_r);
  free(tempImageCol_g);
  free(tempImageCol_b);

  float *imageCol_r = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  float *imageCol_g = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  float *imageCol_b = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  for (int i = 0; i < MATRIX_K * MATRIX_N; i++)
  {
    imageCol_r[i] = (float)imageColChar_r[i];
    imageCol_g[i] = (float)imageColChar_g[i];
    imageCol_b[i] = (float)imageColChar_b[i];
  }

  imageColChar_r = NULL;
  imageColChar_g = NULL;
  imageColChar_b = NULL;
  free(imageColChar_r);
  free(imageColChar_g);
  free(imageColChar_b);

  printf("Apply filters ");
  for (size_t i = 0; i < sizeof(filterIndexes) / sizeof(filterIndexes[0]); i++)
  {
    printf("%s ", filterNames[i]);
  }
  printf("on image with %u x %u pixels for %u iterations\n", image->width, image->height, iterations);

  // Time measurement init
  // Inspired from https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
  hipEvent_t start_time, end_time;
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);

  // Here we do the actual computation!
  // image->data is a 2-dimensional array of pixel which is accessed row first ([y][x])
  // image->rawdata is a 1-dimensional array of pixel containing the same data as image->data
  // each pixel is a struct of 3 unsigned char for the red, blue and green colour channel
  // bmpImage *processImage = newBmpImage(image->width, image->height);

  half *a_fp16_host = (half *)malloc(MATRIX_M * MATRIX_K * sizeof(half));
  half *b_fp16_r_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));
  half *b_fp16_g_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));
  half *b_fp16_b_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));

  printf("To half?\n");
  // Convert float to halves, could also be done more efficiently on the GPU, but this is a simple solution.
  floatToHalf(a_fp16_host, filterCol, MATRIX_M * MATRIX_K);
  floatToHalf(b_fp16_r_host, imageCol_r, MATRIX_K * MATRIX_N);
  floatToHalf(b_fp16_g_host, imageCol_g, MATRIX_K * MATRIX_N);
  floatToHalf(b_fp16_b_host, imageCol_b, MATRIX_K * MATRIX_N);
  printf("To half!\n");

  // All taken from https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/tensor-cores/simpleTensorCoreGEMM.cu
  // float *a_fp32;        // Filter temp
  // float *b_fp32_r;      // Image temp
  // float *b_fp32_g;      // Image temp
  // float *b_fp32_b;      // Image temp
  half *a_fp16;         // Filter
  half *b_fp16_r;       // Image array
  half *b_fp16_g;       // Image array
  half *b_fp16_b;       // Image array
  float *c_wmma_r;      // Device answer array
  float *c_wmma_g;      // Device answer array
  float *c_wmma_b;      // Device answer array
  float *c_host_wmma_r; // Host answer array
  float *c_host_wmma_g; // Host answer array
  float *c_host_wmma_b; // Host answer array

  // cudaErrCheck(hipMalloc((void **)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));

  // cudaErrCheck(hipMalloc((void **)&b_fp32_r, MATRIX_K * MATRIX_N * sizeof(float)));
  // cudaErrCheck(hipMalloc((void **)&b_fp32_g, MATRIX_K * MATRIX_N * sizeof(float)));
  // cudaErrCheck(hipMalloc((void **)&b_fp32_b, MATRIX_K * MATRIX_N * sizeof(float)));

  cudaErrCheck(hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));

  cudaErrCheck(hipMalloc((void **)&b_fp16_r, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void **)&b_fp16_g, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void **)&b_fp16_b, MATRIX_K * MATRIX_N * sizeof(half)));

  cudaErrCheck(hipMalloc((void **)&c_wmma_r, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMalloc((void **)&c_wmma_g, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMalloc((void **)&c_wmma_b, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMemset(c_wmma_r, 0.0f, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMemset(c_wmma_g, 0.0f, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMemset(c_wmma_b, 0.0f, MATRIX_M * MATRIX_N * sizeof(float)));

  c_host_wmma_r = (float *)calloc(sizeof(float), MATRIX_M * MATRIX_N);
  c_host_wmma_g = (float *)calloc(sizeof(float), MATRIX_M * MATRIX_N);
  c_host_wmma_b = (float *)calloc(sizeof(float), MATRIX_M * MATRIX_N);

  printf("Copying over halves?\n");
  cudaErrCheck(hipMemcpy(a_fp16, a_fp16_host, MATRIX_M * MATRIX_K * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_r, b_fp16_r_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_g, b_fp16_g_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_b, b_fp16_b_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  printf("Copying over halves!\n");

  // cudaErrCheck(hipMemcpy(b_fp32_r, imageCol_r, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(b_fp32_g, imageCol_g, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(b_fp32_b, imageCol_b, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));

  // cudaErrCheck(hipDeviceSynchronize());
  filterCol = NULL;
  imageCol_r = NULL;
  imageCol_g = NULL;
  imageCol_b = NULL;
  free(filterCol);
  free(imageCol_r);
  free(imageCol_g);
  free(imageCol_b);

  // curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
  // curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

  // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
  // Convert float to half
  // int filterGridDim = (MATRIX_M * MATRIX_K + 255) / 256;
  // int filterBlockDim = 256;
  // printf("Float to half kernel launch with grid dim %d, block dim %d\n", filterGridDim, filterBlockDim);
  // // convertFp32ToFp16<<<filterGridDim, filterBlockDim>>>(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
  // toHalf(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
  // int imageGridDim = (MATRIX_K * MATRIX_N + 255) / 256;
  // int imageBlockDim = 256;
  // printf("Float to half kernel launch with grid dim %d, block dim %d\n", imageGridDim, imageBlockDim);
  // toHalf(b_fp16_r, b_fp32_r, MATRIX_K * MATRIX_N);
  // toHalf(b_fp16_g, b_fp32_g, MATRIX_K * MATRIX_N);
  // toHalf(b_fp16_b, b_fp32_b, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_r, b_fp32_r, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_g, b_fp32_g, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_b, b_fp32_b, MATRIX_K * MATRIX_N);

  // printf("Before c_wmma copy\n");
  // cudaErrCheck(hipMemcpy(c_wmma_r, c_host_wmma_r, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(c_wmma_g, c_host_wmma_g, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(c_wmma_b, c_host_wmma_b, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // printf("After c_wmma copy\n");

  // curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

  // curandErrCheck(hiprandDestroyGenerator(gen));

  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 16 warps in one block and a block computes a 64x64 output tile
  blockDim.x = 4 * WARP_SIZE;
  blockDim.y = 4;

  gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  if (gridDim.y >= MAX_GRID_DIMENSION)
  {
    // Quick fix
    gridDim.x *= 3;
    gridDim.y = gridDim.y / 3;
  }

  float alpha = 1.0f;
  float beta = 0.0f;

  printf("Launching a kernel with grid dim: %dx%d and block dimension of (%dx%d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

  if (gridDim.x >= MAX_GRID_DIMENSION || gridDim.y >= MAX_GRID_DIMENSION)
  {
    printf("Invalid grid dimensions.\n");
    return 1;
  }

  // Start time measurement
  hipEventRecord(start_time);

  printf("WMMA kernel launch?\n");
  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_r, c_wmma_r, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_g, c_wmma_g, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_b, c_wmma_b, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  cudaErrCheck(hipDeviceSynchronize()); // ? Required?
  printf("WMMA kernel launch!\n");

  // End time measurement
  hipEventRecord(end_time);

  // Check for error
  hipError_t error = hipPeekAtLastError();
  if (error)
  {
    fprintf(stderr, "Error after kernel launch!: %s\n", hipGetErrorString(error));
  }

  printf("Copying to host?\n");
  // We only copy over the stuff we need, which is DESIRED_M * DESIRED_N
  cudaErrCheck(hipMemcpy(c_host_wmma_r, c_wmma_r, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(c_host_wmma_g, c_wmma_g, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  cudaErrCheck(hipMemcpy(c_host_wmma_b, c_wmma_b, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  printf("Copying to host!\n");

  // numberOfFiltersUsed * image->width * image->height == DESIRED_M * DESIRED_N
  pixel *finalImagesRawData = (pixel *)malloc(DESIRED_M * DESIRED_N * sizeof(pixel));
  if (DESIRED_M != numberOfFiltersUsed)
  {
    printf("Invalid DESIRED_M. Aborting.\n");
    return 1;
  }
  if (DESIRED_N != image->width * image->height)
  {
    printf("Invalid DESIRED_N. Aborting.\n");
    return 1;
  }

  for (int m = 0; m < DESIRED_M; m++)
  {
    for (int n = 0; n < DESIRED_N; n++)
    {
      unsigned char r = (unsigned char)c_host_wmma_r[m * DESIRED_N + n];
      unsigned char g = (unsigned char)c_host_wmma_g[m * DESIRED_N + n];
      unsigned char b = (unsigned char)c_host_wmma_b[m * DESIRED_N + n];
      finalImagesRawData[m * DESIRED_N + n] = (pixel){.b = b, .g = g, .r = r};
    }
  }

  // Blocks CPU execution until end_time is recorded
  hipEventSynchronize(end_time);

  float spentTime = 0.0;
  hipEventElapsedTime(&spentTime, start_time, end_time);
  printf("Time spent: %.3f seconds\n", spentTime / 1000);

  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  // Check for error
  error = hipPeekAtLastError();
  if (error)
  {
    fprintf(stderr, "A CUDA error has occurred while cracking: %s\n", hipGetErrorString(error));
  }

  //Write the image back to disk
  // if (saveBmpImage(image, output) != 0)
  // {
  //   fprintf(stderr, "Could not save output to '%s'!\n", output);
  //   freeBmpImage(image);
  //   error_exit(&input, &output);
  // };
  for (int i = 0; i < numberOfFiltersUsed; i++)
  {
    char *outputFilename = (char *)calloc(11, sizeof(char));
    memcpy(image->rawdata, finalImagesRawData + i * DESIRED_N, DESIRED_N * sizeof(pixel));
    sprintf(outputFilename, "img_%d.bmp", filterIndexes[i]);
    if (saveBmpImage(image, outputFilename) != 0)
    {
      fprintf(stderr, "Could not save output to '%s'!\n", outputFilename);
      freeBmpImage(image);
      error_exit(&input, &output);
    };
  }

  graceful_exit(&input, &output);
};
