#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <time.h>

// #include <hip/hip_runtime_api.h> // ? required??
#include <hip/hip_fp16.h>
#include <mma.h> // CUDA WMMA API

using namespace nvcuda; // C++ stuff

extern "C"
{
#include "libs/bitmap.h"
}

#define cudaErrorCheck(ans)               \
  {                                       \
    gpuAssert((ans), __FILE__, __LINE__); \
  }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
  if (code != hipSuccess)
  {
    fprintf(stderr, "GPUassert: %s %s %s %d\n", hipGetErrorName(code), hipGetErrorString(code), file, line);
    if (abort)
      exit(code);
  }
}

// TODO remove one of cudaErr.. or hipError_t..
// Define some error checking macros.
#define cudaErrCheck(stat)                     \
  {                                            \
    cudaErrCheck_((stat), __FILE__, __LINE__); \
  }
void cudaErrCheck_(hipError_t stat, const char *file, int line)
{
  if (stat != hipSuccess)
  {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
  }
}

#define WARP_SIZE 32
#define BLOCK_DIMENSION 16 // A thread block size of 16x16 (256 threads) is a common choice (from https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#thread-hierarchy)

// Convolutional Filter Examples, each with dimension 3,
// gaussian filter with dimension 5

int sobelYFilter[] = {-1, -2, -1,
                      0, 0, 0,
                      1, 2, 1};

int sobelXFilter[] = {-1, -0, 1,
                      -2, 0, 2,
                      -1, 0, 1};

int laplacian1Filter[] = {-1, -4, -1,
                          -4, 20, -4,
                          -1, -4, -1};

int laplacian2Filter[] = {0, 1, 0,
                          1, -4, 1,
                          0, 1, 0};

int laplacian3Filter[] = {-1, -1, -1,
                          -1, 8, -1,
                          -1, -1, -1};

int gaussianFilter[] = {1, 4, 6, 4, 1,
                        4, 16, 24, 16, 4,
                        6, 24, 36, 24, 6,
                        4, 16, 24, 16, 4,
                        1, 4, 6, 4, 1};

const char *filterNames[] = {"SobelY", "SobelX", "Laplacian 1", "Laplacian 2", "Laplacian 3", "Gaussian"};
int *const filters[] = {sobelYFilter, sobelXFilter, laplacian1Filter, laplacian2Filter, laplacian3Filter, gaussianFilter};
unsigned int const filterDims[] = {3, 3, 3, 3, 3, 5};
// float const filterFactors[] = {1.0, 1.0, 1.0, 1.0, 1.0, 1.0 / 256.0}; // TODO ?

const unsigned int MAX_GRID_DIMENSION = 65535;

// const unsigned int numberOfChannels = 3;

// Hardcoded selected filters for now
const unsigned int numberOfFiltersUsed = 5;
const unsigned int filterIndexes[] = {0, 1, 2, 3, 4};
const unsigned int filterDim = 3; // Only one filter dimension supported for now

// WMMA stuff from https://developer.nvidia.com/blog/programming-tensor-cores-cuda-9/
// The only dimensions currently supported by WMMA
const int WMMA_M = 16;
const int WMMA_N = 16;
const int WMMA_K = 16;

// mxk * kxn = mxn
const unsigned int DESIRED_M = numberOfFiltersUsed; // 5 filters
const unsigned int DESIRED_K = 9;                   // 3x3 filters
const unsigned int DESIRED_N = 4000 * 2334;         // 4000x2334 image

// Must be multiplies of 16
const int MATRIX_M = 16;        // I want 5 here (5 filters)
const int MATRIX_K = 16;        // I want 9 here (3x3 filter values)
const int MATRIX_N = DESIRED_N; // It is evenly divisible by 16

// im2col and col2im taken from https://github.com/pluskid/Mocha.jl/blob/master/deps/im2col.cpp#L7
// The function works on one channel at the time because the output image should not be only one channel.
// Furthemore, we assume that we use a valid padding (input dim = output dim) and 1 stride.
void im2col(pixel *img, unsigned char *col, int width, int height, int filterDim, int color)
{
  // The dimension of the col array is im_width*im_height*filter_h*filter_w
  int kernel_h = filterDim,  // Assuming square kernel
      kernel_w = filterDim,  // Assuming square kernel
      pad_h = filterDim / 2, // Same padding
      pad_w = filterDim / 2; // Same padding
  int height_col = (height + 2 * pad_h - kernel_h) + 1;
  int width_col = (width + 2 * pad_w - kernel_w) + 1;
  int channels_col = kernel_h * kernel_w;

  for (int c = 0; c < channels_col; ++c)
  {
    int w_offset = c % kernel_w;
    int h_offset = (c / kernel_w) % kernel_h;
    int c_im = c / (kernel_h * kernel_w);

    for (int h = 0; h < height_col; ++h)
    {
      for (int w = 0; w < width_col; ++w)
      {
        int h_pad = h - pad_h + h_offset;
        int w_pad = w - pad_w + w_offset;
        int index_col = (c * height_col + h) * width_col + w;
        int index_im = (c_im * height + h_pad) * width + w_pad;
        if (h_pad >= 0 && h_pad < height && w_pad >= 0 && w_pad < width)
        {
          if (color == 0)
          {
            col[index_col] = img[index_im].r;
          }
          else if (color == 1)
          {
            col[index_col] = img[index_im].g;
          }
          else
          {
            col[index_col] = img[index_im].b;
          }
        }
        else
        {
          col[index_col] = 0;
        }
      }
    }
  }
}

void buildFilterArray(float *array)
{
  // The filter is of dimension mxk
  // The 0s padding is included in this function
  for (int i = 0; i < MATRIX_M; i++)
  {
    for (int j = 0; j < MATRIX_K; j++)
    {
      int arrayIndex = i * MATRIX_K + j;
      if (i < DESIRED_M && j < DESIRED_K)
      {
        unsigned int filterIndex = filterIndexes[i];
        array[arrayIndex] = (float)filters[filterIndex][j];
      }
      else
      {
        array[arrayIndex] = 0.0f;
      }
    }
  }
}

void buildImageArray(unsigned char *outputCol, unsigned char *inputCol, unsigned int intputColLength)
{
  // The image is of dimension kxn
  // This function adds the 5 0s to the colums
  memcpy(outputCol, inputCol, intputColLength * sizeof(unsigned char));
  for (int k = 0; k < MATRIX_K - DESIRED_K; k++)
  {
    for (int n = 0; n < MATRIX_N; n++)
    {
      outputCol[intputColLength + k * MATRIX_N + n] = 0;
    }
  }
}

void cleanup(char **input, char **output)
{
  if (*input)
    free(*input);
  if (*output)
    free(*output);
}

void graceful_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(0);
}

void error_exit(char **input, char **output)
{
  cleanup(input, output);
  exit(1);
}

// Helper function to swap bmpImageChannel pointers

void swapImageRawdata(pixel **one, pixel **two)
{
  pixel *helper = *two;
  *two = *one;
  *one = helper;
}

void swapImage(bmpImage **one, bmpImage **two)
{
  bmpImage *helper = *two;
  *two = *one;
  *one = helper;
}

// This function is only used for creating the baseline images for filter 0-4
// Apply convolutional filter on image data with the use of shared memory
__global__ void apply_filter(pixel *out, pixel *in, unsigned int width, unsigned int height, int *filter, unsigned int filterDim, float filterFactor)
{

  // Now instead of using the filter directly from global memory, we want to copy the filter to shared memory.
  // Dynamic shared memory because the filterDim is not known at compile time.

  // This one holds all of the data
  extern __shared__ int s[];

  int *shared_filter = s;                                                // Length of filterDim * filterDim
  pixel *shared_pixels = (pixel *)&shared_filter[filterDim * filterDim]; // Length of BLOCK_DIMENSION * BLOCK_DIMENSION

  for (int i = 0; i < filterDim * filterDim; i++)
  {
    shared_filter[i] = filter[i];
  }

  // Sync to make sure that all threads have completed the loads to shared memory
  __syncthreads();
  // Now we can use shared_filter!

  // Because shared memory is only shared between blocks, it makes sense to make the shared memory array for
  // the image as big as the block, since each thread in the block changes one pixel.

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;

  // Handle out of bounds
  if (x >= width || y >= height)
  {
    // __syncthreads(); // ? Needed? Think so, to avoid deadlock
    return;
  }

  // Set the position in the block to the correct value
  shared_pixels[threadIdx.y * BLOCK_DIMENSION + threadIdx.x] = in[y * width + x];

  // Sync to make sure that all threads have completed the loads to shared memory
  __syncthreads();
  // Now we can use shared_pixels!

  unsigned int const filterCenter = (filterDim / 2);
  int ar = 0, ag = 0, ab = 0;
  for (unsigned int ky = 0; ky < filterDim; ky++)
  {
    int nky = filterDim - 1 - ky;
    for (unsigned int kx = 0; kx < filterDim; kx++)
    {
      int nkx = filterDim - 1 - kx;

      int yy = y + (ky - filterCenter);
      int xx = x + (kx - filterCenter);

      // Now, since the edge threads needs pixels outside the block's shared memory,
      // we need to check its position.

      if (xx >= 0 && xx < BLOCK_DIMENSION && yy >= 0 && yy < BLOCK_DIMENSION)
      {
        ar += shared_pixels[yy * BLOCK_DIMENSION + xx].r * shared_filter[nky * filterDim + nkx];
        ag += shared_pixels[yy * BLOCK_DIMENSION + xx].g * shared_filter[nky * filterDim + nkx];
        ab += shared_pixels[yy * BLOCK_DIMENSION + xx].b * shared_filter[nky * filterDim + nkx];
      }
      // Else if the normal code from task 1-4
      else if (xx >= 0 && xx < (int)width && yy >= 0 && yy < (int)height)
      {
        ar += in[yy * width + xx].r * shared_filter[nky * filterDim + nkx];
        ag += in[yy * width + xx].g * shared_filter[nky * filterDim + nkx];
        ab += in[yy * width + xx].b * shared_filter[nky * filterDim + nkx];
      }
    }
  }

  ar *= filterFactor;
  ag *= filterFactor;
  ab *= filterFactor;

  ar = (ar < 0) ? 0 : ar;
  ag = (ag < 0) ? 0 : ag;
  ab = (ab < 0) ? 0 : ab;

  out[y * width + x].r = (ar > 255) ? 255 : ar;
  out[y * width + x].g = (ag > 255) ? 255 : ag;
  out[y * width + x].b = (ab > 255) ? 255 : ab;
}

// __global__ void apply_filter_GEMM(bmpImage *out, bmpImage *in, int *filters, int numberOfFilters, unsigned int filterDim, unsigned int filterSize, float filterFactor)
__global__ void apply_filter_GEMM(half *a, half *b, float *c, int M, int N, int K, float alpha, float beta)
{
  // Leading dimensions. Packed with no transpositions.
  int lda = M;
  int ldb = K;
  int ldc = M;

  // From https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#wmma-example
  // Tile using a 2D grid
  int warpM = (blockIdx.x * blockDim.x + threadIdx.x) / WARP_SIZE;
  int warpN = (blockIdx.y * blockDim.y + threadIdx.y);

  // Declare the fragments
  wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> a_frag;
  wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag; // This holds A*B
  wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;

  // Initialize the output to zero
  wmma::fill_fragment(acc_frag, 0.0f);

  // A*B
  // Loop over the K-dimension
  for (int i = 0; i < K; i += WMMA_K)
  {
    int aRow = warpM * WMMA_M;
    int aCol = i;
    int bRow = i;
    int bCol = warpN * WMMA_N;

    // Bounds checking
    if (aRow < M && aCol < K && bRow < K && bCol < N)
    {
      // Load the inputs
      wmma::load_matrix_sync(a_frag, a + aRow + aCol * lda, lda);
      wmma::load_matrix_sync(b_frag, b + bRow + bCol * ldb, ldb);

      // Perform the matrix multiplication
      wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }
  }

  // AB + C
  // Load in the current value of c, scale it by beta, and add this our result scaled by alpha
  int cRow = warpM * WMMA_M;
  int cCol = warpN * WMMA_N;

  if (cRow < M && cCol < N)
  {
    wmma::load_matrix_sync(c_frag, c + cRow + cCol * ldc, ldc, wmma::mem_col_major);

    for (int i = 0; i < c_frag.num_elements; i++)
    {
      c_frag.x[i] = alpha * acc_frag.x[i] + beta * c_frag.x[i];
    }

    // Store the output
    wmma::store_matrix_sync(c + cRow + cCol * ldc, c_frag, ldc, wmma::mem_col_major);
  }
}

__host__ void floatToHalf(half *out, float *in, int n)
{
  for (int i = 0; i < n; i++)
  {
    out[i] = __float2half(in[i]);
  }
}

// __global__ void convertFp32ToFp16(half *out, float *in, int n)
// {
//   int idx = blockDim.x * blockIdx.x + threadIdx.x;
//   if (idx < n)
//   {
//     out[idx] = in[idx];
//   }
// }

void help(char const *exec, char const opt, char const *optarg)
{
  FILE *out = stdout;
  if (opt != 0)
  {
    out = stderr;
    if (optarg)
    {
      fprintf(out, "Invalid parameter - %c %s\n", opt, optarg);
    }
    else
    {
      fprintf(out, "Invalid parameter - %c\n", opt);
    }
  }
  fprintf(out, "%s [options] <input-bmp> <output-bmp>\n", exec);
  fprintf(out, "\n");
  fprintf(out, "Options:\n");
  fprintf(out, "  -i, --iterations <iterations>    number of iterations (1)\n");

  fprintf(out, "\n");
  fprintf(out, "Example: %s before.bmp after.bmp -i 10000\n", exec);
}

int main(int argc, char **argv)
{
  /*
    Parameter parsing, don't change this!
   */
  unsigned int iterations = 1;
  char *output = NULL;
  char *input = NULL;

  static struct option const long_options[] = {
      {"help", no_argument, 0, 'h'},
      {"iterations", required_argument, 0, 'i'},
      {0, 0, 0, 0}};

  static char const *short_options = "h:i:";
  {
    char *endptr;
    int c;
    // int parse;
    int option_index = 0;
    while ((c = getopt_long(argc, argv, short_options, long_options, &option_index)) != -1)
    {
      switch (c)
      {
      case 'h':
        help(argv[0], 0, NULL);
        graceful_exit(&input, &output);
      case 'i':
        iterations = strtol(optarg, &endptr, 10);
        if (endptr == optarg)
        {
          help(argv[0], c, optarg);
          error_exit(&input, &output);
        }
        break;
      default:
        abort();
      }
    }
  }

  if (argc <= (optind + 1))
  {
    help(argv[0], ' ', "Not enough arugments");
    error_exit(&input, &output);
  }

  unsigned int arglen = strlen(argv[optind]);
  input = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(input, argv[optind], arglen);
  optind++;

  arglen = strlen(argv[optind]);
  output = (char *)calloc(arglen + 1, sizeof(char));
  strncpy(output, argv[optind], arglen);
  optind++;

  /*
    End of Parameter parsing!
   */

  /*
    Create the BMP image and load it from disk.
   */
  bmpImage *image = newBmpImage(0, 0);
  if (image == NULL)
  {
    fprintf(stderr, "Could not allocate new image!\n");
    error_exit(&input, &output);
  }

  if (loadBmpImage(image, input) != 0)
  {
    fprintf(stderr, "Could not load bmp image '%s'!\n", input);
    freeBmpImage(image);
    error_exit(&input, &output);
  }

  // Const initialization
  for (unsigned int i = 1; i < numberOfFiltersUsed; i++)
  {
    if (filterDims[filterIndexes[i]] != filterDim)
    {
      printf("Unequal filter dimension used");
      exit(1);
    }
  }
  // const float usedFilterFactor = filterFactors[filterIndexes[0]]; // TODO

  float *filterCol = (float *)malloc(MATRIX_M * MATRIX_K * sizeof(float));
  buildFilterArray(filterCol);

  unsigned int tempImageColLength = image->width * image->height * filterDim * filterDim;
  unsigned char *tempImageCol_r = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  unsigned char *tempImageCol_g = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  unsigned char *tempImageCol_b = (unsigned char *)malloc(tempImageColLength * sizeof(unsigned char));
  im2col(image->rawdata, tempImageCol_r, image->width, image->height, filterDim, 0);
  im2col(image->rawdata, tempImageCol_g, image->width, image->height, filterDim, 1);
  im2col(image->rawdata, tempImageCol_b, image->width, image->height, filterDim, 2);

  unsigned char *imageColChar_r = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  unsigned char *imageColChar_g = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  unsigned char *imageColChar_b = (unsigned char *)malloc(MATRIX_K * MATRIX_N * sizeof(unsigned char));
  buildImageArray(imageColChar_r, tempImageCol_r, tempImageColLength);
  buildImageArray(imageColChar_g, tempImageCol_g, tempImageColLength);
  buildImageArray(imageColChar_b, tempImageCol_b, tempImageColLength);

  // TOOD try to fix these, it crashes on free _b
  tempImageCol_r = NULL;
  tempImageCol_g = NULL;
  tempImageCol_b = NULL;
  free(tempImageCol_r);
  free(tempImageCol_g);
  free(tempImageCol_b);

  float *imageCol_r = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  float *imageCol_g = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  float *imageCol_b = (float *)malloc(MATRIX_K * MATRIX_N * sizeof(float));
  for (int i = 0; i < MATRIX_K * MATRIX_N; i++)
  {
    imageCol_r[i] = (float)imageColChar_r[i];
    imageCol_g[i] = (float)imageColChar_g[i];
    imageCol_b[i] = (float)imageColChar_b[i];
  }

  imageColChar_r = NULL;
  imageColChar_g = NULL;
  imageColChar_b = NULL;
  free(imageColChar_r);
  free(imageColChar_g);
  free(imageColChar_b);

  printf("Apply filters ");
  for (size_t i = 0; i < sizeof(filterIndexes) / sizeof(filterIndexes[0]); i++)
  {
    printf("%s ", filterNames[i]);
  }
  printf("on image with %u x %u pixels for %u iterations\n", image->width, image->height, iterations);

  // Time measurement init
  // Inspired from https://developer.nvidia.com/blog/how-implement-performance-metrics-cuda-cc/
  hipEvent_t start_time, end_time;
  hipEventCreate(&start_time);
  hipEventCreate(&end_time);

  // Here we do the actual computation!
  // image->data is a 2-dimensional array of pixel which is accessed row first ([y][x])
  // image->rawdata is a 1-dimensional array of pixel containing the same data as image->data
  // each pixel is a struct of 3 unsigned char for the red, blue and green colour channel
  // bmpImage *processImage = newBmpImage(image->width, image->height);

  // int image_size = image->width * image->height * sizeof(pixel);

  // We could also made all filters __device__ available, but it is simple to copy over only the needed one
  // pixel *d_image_rawdata, *d_process_image_rawdata;
  // int *d_filters;

  // hipMalloc((void **)&d_image_rawdata, image_size); // ! * numberOfFilters ?
  // hipMalloc((void **)&d_process_image_rawdata, image_size * numberOfFiltersUsed);
  // hipMalloc((void **)&d_filters, filter_size * numberOfFiltersUsed);

  // hipMemcpy(d_image_rawdata, image->rawdata, image_size, hipMemcpyHostToDevice);

  // We allocate one thread per pixel
  // gridSize and blockSize inspired from Section 2.2. in the CUDA Programming Guide
  // dim3 blockSize(BLOCK_DIMENSION, BLOCK_DIMENSION); // Threads per block
  // printf("The grid has thread blocks of dimension (%d width * %d height)\n", blockSize.x, blockSize.y);

  // We may need to add 1 extra block to width or height if the image's dimensions are not evenly divided by the block's dimension
  // int extraWidth = 0;
  // int extraHeight = 0;
  //
  // if (image->width % blockSize.x != 0)
  // {
  //   extraWidth = 1;
  // }
  // if (image->height % blockSize.y != 0)
  // {
  //   extraHeight = 1;
  // }
  // dim3 gridSize(image->width / blockSize.x + extraWidth, image->height / blockSize.y + extraHeight); // Number of blocks
  // printf("Launching a grid of dimension (%d width * %d height)\n", image->width / blockSize.x + extraWidth, image->height / blockSize.y + extraHeight);

  half *a_fp16_host = (half *)malloc(MATRIX_M * MATRIX_K * sizeof(half));
  half *b_fp16_r_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));
  half *b_fp16_g_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));
  half *b_fp16_b_host = (half *)malloc(MATRIX_K * MATRIX_N * sizeof(half));

  // All taken from https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/tensor-cores/simpleTensorCoreGEMM.cu
  // float *a_fp32;        // Filter temp
  // float *b_fp32_r;      // Image temp
  // float *b_fp32_g;      // Image temp
  // float *b_fp32_b;      // Image temp
  half *a_fp16;         // Filter
  half *b_fp16_r;       // Image array
  half *b_fp16_g;       // Image array
  half *b_fp16_b;       // Image array
  float *c_wmma_r;      // Device answer array
  float *c_wmma_g;      // Device answer array
  float *c_wmma_b;      // Device answer array
  float *c_host_wmma_r; // Host answer array
  float *c_host_wmma_g; // Host answer array
  float *c_host_wmma_b; // Host answer array

  printf("To half?\n");
  floatToHalf(a_fp16_host, filterCol, MATRIX_M * MATRIX_K);
  floatToHalf(b_fp16_r_host, imageCol_r, MATRIX_K * MATRIX_N);
  floatToHalf(b_fp16_g_host, imageCol_g, MATRIX_K * MATRIX_N);
  floatToHalf(b_fp16_b_host, imageCol_b, MATRIX_K * MATRIX_N);
  printf("To half!\n");

  // cudaErrCheck(hipMalloc((void **)&a_fp32, MATRIX_M * MATRIX_K * sizeof(float)));

  // cudaErrCheck(hipMalloc((void **)&b_fp32_r, MATRIX_K * MATRIX_N * sizeof(float)));
  // cudaErrCheck(hipMalloc((void **)&b_fp32_g, MATRIX_K * MATRIX_N * sizeof(float)));
  // cudaErrCheck(hipMalloc((void **)&b_fp32_b, MATRIX_K * MATRIX_N * sizeof(float)));

  cudaErrCheck(hipMalloc((void **)&a_fp16, MATRIX_M * MATRIX_K * sizeof(half)));

  cudaErrCheck(hipMalloc((void **)&b_fp16_r, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void **)&b_fp16_g, MATRIX_K * MATRIX_N * sizeof(half)));
  cudaErrCheck(hipMalloc((void **)&b_fp16_b, MATRIX_K * MATRIX_N * sizeof(half)));

  cudaErrCheck(hipMalloc((void **)&c_wmma_r, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMalloc((void **)&c_wmma_g, MATRIX_M * MATRIX_N * sizeof(float)));
  cudaErrCheck(hipMalloc((void **)&c_wmma_b, MATRIX_M * MATRIX_N * sizeof(float)));

  c_host_wmma_r = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));
  c_host_wmma_g = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));
  c_host_wmma_b = (float *)malloc(MATRIX_M * MATRIX_N * sizeof(float));

  printf("Copying over halves?\n");
  cudaErrCheck(hipMemcpy(a_fp16, a_fp16_host, MATRIX_M * MATRIX_K * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_r, b_fp16_r_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_g, b_fp16_g_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  cudaErrCheck(hipMemcpy(b_fp16_b, b_fp16_b_host, MATRIX_K * MATRIX_N * sizeof(half), hipMemcpyHostToDevice));
  printf("Copying over halves!\n");

  // curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
  // curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

  // Fill device memory with filter col and image col
  // cudaErrCheck(hipMemcpy(a_fp32, filterCol, MATRIX_M * MATRIX_K * sizeof(float), hipMemcpyHostToDevice));

  // cudaErrCheck(hipMemcpy(b_fp32_r, imageCol_r, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(b_fp32_g, imageCol_g, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(b_fp32_b, imageCol_b, MATRIX_K * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));

  // cudaErrCheck(hipDeviceSynchronize());
  filterCol = NULL;
  imageCol_r = NULL;
  imageCol_g = NULL;
  imageCol_b = NULL;
  free(filterCol);
  free(imageCol_r);
  free(imageCol_g);
  free(imageCol_b);

  // curandErrCheck(hiprandGenerateUniform(gen, a_fp32, MATRIX_M * MATRIX_K));
  // curandErrCheck(hiprandGenerateUniform(gen, b_fp32, MATRIX_K * MATRIX_N));

  // hiprand doesn't currently support fp16 so we generate in fp32 and convert to fp16.
  // Convert float to half
  // int filterGridDim = (MATRIX_M * MATRIX_K + 255) / 256;
  // int filterBlockDim = 256;
  // printf("Float to half kernel launch with grid dim %d, block dim %d\n", filterGridDim, filterBlockDim);
  // // convertFp32ToFp16<<<filterGridDim, filterBlockDim>>>(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
  // toHalf(a_fp16, a_fp32, MATRIX_M * MATRIX_K);
  // int imageGridDim = (MATRIX_K * MATRIX_N + 255) / 256;
  // int imageBlockDim = 256;
  // printf("Float to half kernel launch with grid dim %d, block dim %d\n", imageGridDim, imageBlockDim);
  // toHalf(b_fp16_r, b_fp32_r, MATRIX_K * MATRIX_N);
  // toHalf(b_fp16_g, b_fp32_g, MATRIX_K * MATRIX_N);
  // toHalf(b_fp16_b, b_fp32_b, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_r, b_fp32_r, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_g, b_fp32_g, MATRIX_K * MATRIX_N);
  // convertFp32ToFp16<<<imageGridDim, imageBlockDim>>>(b_fp16_b, b_fp32_b, MATRIX_K * MATRIX_N);

  printf("Before sync\n");
  cudaErrCheck(hipDeviceSynchronize()); // ? Required?
  printf("After sync\n");

  // // ? Is this required?
  // for (int i = 0; i < MATRIX_M * MATRIX_N; i++)
  // {
  //   c_host_wmma_r[i] = 0.0f;
  // }
  // printf("red done\n");
  // for (int i = 0; i < MATRIX_M * MATRIX_N; i++)
  // {
  //   c_host_wmma_g[i] = 0.0f;
  // }
  // printf("g done\n");
  // for (int i = 0; i < MATRIX_M * MATRIX_N; i++)
  // {
  //   c_host_wmma_b[i] = 0.0f;
  // }
  // printf("b done\n");

  // printf("Before c_wmma copy\n");
  // cudaErrCheck(hipMemcpy(c_wmma_r, c_host_wmma_r, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(c_wmma_g, c_host_wmma_g, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // cudaErrCheck(hipMemcpy(c_wmma_b, c_host_wmma_b, MATRIX_M * MATRIX_N * sizeof(float), hipMemcpyHostToDevice));
  // printf("After c_wmma copy\n");

  // curandErrCheck(hiprandGenerateUniform(gen, c, MATRIX_M * MATRIX_N));

  // curandErrCheck(hiprandDestroyGenerator(gen));

  dim3 gridDim;
  dim3 blockDim;

  // blockDim.x must be a multple of warpSize
  // 16 warps in one block and a block computes a 64x64 output tile
  blockDim.x = 4 * WARP_SIZE;
  blockDim.y = 4;

  gridDim.x = (MATRIX_M + (WMMA_M * blockDim.x / 32 - 1)) / (WMMA_M * blockDim.x / 32);
  gridDim.y = (MATRIX_N + WMMA_N * blockDim.y - 1) / (WMMA_N * blockDim.y);

  // TODO remove
  float alpha = 1.0f;
  float beta = 1.0f;

  printf("Launching a kernel with grid dim: %dx%d and block dimension of (%dx%d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);

  if (gridDim.y >= MAX_GRID_DIMENSION)
  {
    // TODO fix
    gridDim.x *= 3;
    gridDim.y = gridDim.y / 3;
    printf("INSTEAD: Launching a kernel with grid dim: %dx%d and block dimension of (%dx%d)\n", gridDim.x, gridDim.y, blockDim.x, blockDim.y);
  }

  if (gridDim.x >= MAX_GRID_DIMENSION || gridDim.y >= MAX_GRID_DIMENSION)
  {
    printf("Invalid grid dimensions.\n");
    return 1;
  }

  // Start time measurement
  hipEventRecord(start_time);

  // for (unsigned int i = 0; i < iterations; i++)
  // {
  // int sharedMemoryUsedPerBlock = numberOfFiltersUsed * usedFilterDimension * usedFilterDimension * sizeof(int) + BLOCK_DIMENSION * BLOCK_DIMENSION * sizeof(pixel);
  // apply_filter<<<gridSize, blockSize, sharedMemoryUsedPerBlock>>>(
  //     d_process_image_rawdata, // Out
  //     d_image_rawdata,         // In
  //     image->width,
  //     image->height,
  //     // filters[filterIndex],
  //     d_filters,
  //     numberOfFiltersUsed,
  //     usedFilterDimension,
  //     filter_size,
  //     usedFilterFactor);

  printf("WMMA kernel launch?\n");

  // {

  //   bool empty = true;
  //   for (int i = 0; i < MATRIX_M * MATRIX_K; i++)
  //   {
  //     if (a_fp16[i] != 0)
  //     {
  //       printf("%d\n", a_fp16[i]);
  //       empty = false;
  //     }
  //   }
  //   if (empty)
  //   {
  //     printf("Filter col is all 0s\n");
  //     return 1;
  //   }
  // }

  // {

  //   bool empty = true;
  //   for (int i = 0; i < MATRIX_K * MATRIX_N; i++)
  //   {
  //     if (b_fp16_r[i] != 0 || b_fp16_g[i] == 0 || b_fp16_b[i] == 0)
  //     {
  //       printf("%d, %d, %d\n", b_fp16_r[i], b_fp16_g[i], b_fp16_b[i]);
  //       empty = false;
  //     }
  //   }
  //   if (empty)
  //   {
  //     printf("Image col is all 0s\n");
  //     return 1;
  //   }
  // }

  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_r, c_wmma_r, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_g, c_wmma_g, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  apply_filter_GEMM<<<gridDim, blockDim>>>(a_fp16, b_fp16_b, c_wmma_b, MATRIX_M, MATRIX_N, MATRIX_K, alpha, beta);
  cudaErrCheck(hipDeviceSynchronize()); // ? Required?
  printf("WMMA kernel launch!\n");
  // swapImage(&processImage, &image);
  // swapImageRawdata(&d_process_image_rawdata, &d_image_rawdata);
  // }

  // Check for error
  hipError_t error = hipPeekAtLastError();
  if (error)
  {
    fprintf(stderr, "Error after kernel launch!: %s\n", hipGetErrorString(error));
  }

  // End time measurement
  hipEventRecord(end_time);

  printf("Copying to host?\n");
  // We only copy over the stuff we need, which is DESIRED_M * DESIRED_N
  cudaErrCheck(hipMemcpy(c_host_wmma_r, c_wmma_r, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  // printf("Copying to host?\n");
  cudaErrCheck(hipMemcpy(c_host_wmma_g, c_wmma_g, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  // printf("Copying to host?\n");
  cudaErrCheck(hipMemcpy(c_host_wmma_b, c_wmma_b, DESIRED_M * DESIRED_N * sizeof(float), hipMemcpyDeviceToHost));
  printf("Copying to host!\n");

  // TODO clean this lol
  pixel *finalImageRawData0 = (pixel *)malloc(image->width * image->height * sizeof(pixel));
  pixel *finalImageRawData1 = (pixel *)malloc(image->width * image->height * sizeof(pixel));
  pixel *finalImageRawData2 = (pixel *)malloc(image->width * image->height * sizeof(pixel));
  pixel *finalImageRawData3 = (pixel *)malloc(image->width * image->height * sizeof(pixel));
  pixel *finalImageRawData4 = (pixel *)malloc(image->width * image->height * sizeof(pixel));
  if (DESIRED_N != image->width * image->height)
  {
    printf("wtf\n");
    return 1;
  }
  for (int m = 0; m < DESIRED_M; m++)
  {
    for (int n = 0; n < DESIRED_N; n++)
    {
      unsigned char r = (unsigned char)c_host_wmma_r[m * DESIRED_N + n];
      unsigned char g = (unsigned char)c_host_wmma_g[m * DESIRED_N + n];
      unsigned char b = (unsigned char)c_host_wmma_b[m * DESIRED_N + n];
      pixel p = (pixel){.b = b, .g = g, .r = r}; // !
      switch (m)
      {
      case 0:
        finalImageRawData0[n] = p;
        break;
      case 1:
        finalImageRawData1[n] = p;
        break;
      case 2:
        finalImageRawData2[n] = p;
        break;
      case 3:
        finalImageRawData3[n] = p;
        break;
      case 4:
        finalImageRawData4[n] = p;
        break;
      default:
        break;
      }
    }
  }

  // hipMemcpy(image->rawdata, d_image_rawdata, image_size, hipMemcpyDeviceToHost);

  // hipFree(d_image_rawdata);
  // hipFree(d_process_image_rawdata);

  // Blocks CPU execution until end_time is recorded
  hipEventSynchronize(end_time);

  float spentTime = 0.0;
  hipEventElapsedTime(&spentTime, start_time, end_time);
  printf("Time spent: %.3f seconds\n", spentTime / 1000);

  hipEventDestroy(start_time);
  hipEventDestroy(end_time);

  // Check for error
  error = hipPeekAtLastError();
  if (error)
  {
    fprintf(stderr, "A CUDA error has occurred while cracking: %s\n", hipGetErrorString(error));
  }

  //Write the image back to disk
  // if (saveBmpImage(image, output) != 0)
  // {
  //   fprintf(stderr, "Could not save output to '%s'!\n", output);
  //   freeBmpImage(image);
  //   error_exit(&input, &output);
  // };
  memcpy(image->rawdata, finalImageRawData0, image->width * image->height * sizeof(pixel));
  char outputName[11] = "wmma_x.bmp";
  strcpy(outputName, "wmma_0.bmp");
  if (saveBmpImage(image, outputName) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", outputName);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  memcpy(image->rawdata, finalImageRawData1, image->width * image->height * sizeof(pixel));
  strcpy(outputName, "wmma_1.bmp");
  if (saveBmpImage(image, outputName) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", outputName);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  memcpy(image->rawdata, finalImageRawData2, image->width * image->height * sizeof(pixel));
  strcpy(outputName, "wmma_2.bmp");
  if (saveBmpImage(image, outputName) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", outputName);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  memcpy(image->rawdata, finalImageRawData3, image->width * image->height * sizeof(pixel));
  strcpy(outputName, "wmma_3.bmp");
  if (saveBmpImage(image, outputName) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", outputName);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  memcpy(image->rawdata, finalImageRawData4, image->width * image->height * sizeof(pixel));
  strcpy(outputName, "wmma_4.bmp");
  if (saveBmpImage(image, outputName) != 0)
  {
    fprintf(stderr, "Could not save output to '%s'!\n", outputName);
    freeBmpImage(image);
    error_exit(&input, &output);
  };

  graceful_exit(&input, &output);
};
